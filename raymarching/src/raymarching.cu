#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

// #include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")



inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabs(y), fabs(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
	uint32_t xx = __expand_bits(x);
	uint32_t yy = __expand_bits(y);
	uint32_t zz = __expand_bits(z);
	return xx | (yy << 1) | (zz << 2);
}

inline __host__ __device__ uint32_t __morton3D_invert(uint32_t x)
{
	x = x & 0x49249249;
	x = (x | (x >> 2)) & 0xc30c30c3;
	x = (x | (x >> 4)) & 0x0f00f00f;
	x = (x | (x >> 8)) & 0xff0000ff;
	x = (x | (x >> 16)) & 0x0000ffff;
	return x;
}


////////////////////////////////////////////////////
/////////////           utils          /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// nears/fars: [N]
// scalar_t should always be float in use.
template <typename scalar_t>
__global__ void kernel_near_far_from_aabb(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const scalar_t * __restrict__ aabb,
    const uint32_t N,
    const float min_near,
    scalar_t * nears, scalar_t * fars
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // get near far (assume cube scene)
    float near = (aabb[0] - ox) * rdx;
    float far = (aabb[3] - ox) * rdx;
    if (near > far) swapf(near, far);

    float near_y = (aabb[1] - oy) * rdy;
    float far_y = (aabb[4] - oy) * rdy;
    if (near_y > far_y) swapf(near_y, far_y);

    if (near > far_y || near_y > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_y > near) near = near_y;
    if (far_y < far) far = far_y;

    float near_z = (aabb[2] - oz) * rdz;
    float far_z = (aabb[5] - oz) * rdz;
    if (near_z > far_z) swapf(near_z, far_z);

    if (near > far_z || near_z > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_z > near) near = near_z;
    if (far_z < far) far = far_z;

    if (near < min_near) near = min_near;

    nears[n] = near;
    fars[n] = far;
}


void near_far_from_aabb(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor aabb, const uint32_t N, const float min_near, at::Tensor nears, at::Tensor fars) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "near_far_from_aabb", ([&] {
        kernel_near_far_from_aabb<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), aabb.data_ptr<scalar_t>(), N, min_near, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>());
    }));
}


// rays_o/d: [N, 3]
// radius: float
// coords: [N, 2]
template <typename scalar_t>
__global__ void kernel_sph_from_ray(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const float radius,
    const uint32_t N,
    scalar_t * coords
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    coords += n * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // solve t from || o + td || = radius
    const float A = dx * dx + dy * dy + dz * dz;
    const float B = ox * dx + oy * dy + oz * dz; // in fact B / 2
    const float C = ox * ox + oy * oy + oz * oz - radius * radius;

    const float t = (- B + sqrtf(B * B - A * C)) / A; // always use the larger solution (positive)

    // solve theta, phi (assume y is the up axis)
    const float x = ox + t * dx, y = oy + t * dy, z = oz + t * dz;
    const float theta = atan2(sqrtf(x * x + z * z), y); // [0, PI)
    const float phi = atan2(z, x); // [-PI, PI)

    // normalize to [-1, 1]
    coords[0] = 2 * theta * RPI() - 1;
    coords[1] = phi * RPI();
}


void sph_from_ray(const at::Tensor rays_o, const at::Tensor rays_d, const float radius, const uint32_t N, at::Tensor coords) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "sph_from_ray", ([&] {
        kernel_sph_from_ray<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), radius, N, coords.data_ptr<scalar_t>());
    }));
}


// coords: int32, [N, 3]
// indices: int32, [N]
__global__ void kernel_morton3D(
    const int * __restrict__ coords,
    const uint32_t N,
    int * indices
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;
    indices[n] = __morton3D(coords[0], coords[1], coords[2]);
}


void morton3D(const at::Tensor coords, const uint32_t N, at::Tensor indices) {
    static constexpr uint32_t N_THREAD = 256;
    kernel_morton3D<<<div_round_up(N, N_THREAD), N_THREAD>>>(coords.data_ptr<int>(), N, indices.data_ptr<int>());
}


// indices: int32, [N]
// coords: int32, [N, 3]
__global__ void kernel_morton3D_invert(
    const int * __restrict__ indices,
    const uint32_t N,
    int * coords
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;

    const int ind = indices[n];

    coords[0] = __morton3D_invert(ind >> 0);
    coords[1] = __morton3D_invert(ind >> 1);
    coords[2] = __morton3D_invert(ind >> 2);
}


void morton3D_invert(const at::Tensor indices, const uint32_t N, at::Tensor coords) {
    static constexpr uint32_t N_THREAD = 256;
    kernel_morton3D_invert<<<div_round_up(N, N_THREAD), N_THREAD>>>(indices.data_ptr<int>(), N, coords.data_ptr<int>());
}


// grid: float, [C, H, H, H]
// N: int, C * H * H * H / 8
// density_thresh: float
// bitfield: uint8, [N]
template <typename scalar_t>
__global__ void kernel_packbits(
    const scalar_t * __restrict__ grid,
    const uint32_t N,
    const float density_thresh,
    uint8_t * bitfield
) {
    // parallel per byte
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    grid += n * 8;

    uint8_t bits = 0;

    #pragma unroll
    for (uint8_t i = 0; i < 8; i++) {
        bits |= (grid[i] > density_thresh) ? ((uint8_t)1 << i) : 0;
    }

    bitfield[n] = bits;
}


void packbits(const at::Tensor grid, const uint32_t N, const float density_thresh, at::Tensor bitfield) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grid.scalar_type(), "packbits", ([&] {
        kernel_packbits<<<div_round_up(N, N_THREAD), N_THREAD>>>(grid.data_ptr<scalar_t>(), N, density_thresh, bitfield.data_ptr<uint8_t>());
    }));
}

////////////////////////////////////////////////////
/////////////         training         /////////////
////////////////////////////////////////////////////


template <typename scalar_t>
__global__ void kernel_march_rays_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,

    const scalar_t * __restrict__ sun_d,
    const scalar_t * __restrict__ ts,

    const uint8_t * __restrict__ grid,
    const float bound,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M,
    const scalar_t* __restrict__ nears,
    const scalar_t* __restrict__ fars,
    scalar_t * xyzs, scalar_t * dirs, scalar_t * deltas,

    scalar_t * sun_dirs,
    scalar_t * all_ts,

    int * rays,
    int * counter,
    const scalar_t* __restrict__ noises
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    sun_d += n * 3;
    ts += n * 4;

    // ray marching
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];

    const float sdx = sun_d[0], sdy = sun_d[1], sdz = sun_d[2];
    const float tsx = ts[0], tsy = ts[1], tsz = ts[2], tsw = ts[3];

    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;

    const float near = nears[n];
    const float far = fars[n];
    const float noise = noises[n];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;

    float t0 = near;

    // perturb
    t0 += clamp(t0 * dt_gamma, dt_min, dt_max) * noise;

    // first pass: estimation of num_steps
    float t = t0;
    uint32_t num_steps = 0;

    //if (t < far) printf("valid ray %d t=%f near=%f far=%f \n", n, t, near, far);

    while (t < far && num_steps < max_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;

        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        //if (n == 0) printf("t=%f density=%f vs thresh=%f step=%d\n", t, density, density_thresh, num_steps);

        if (occ) {
            num_steps++;
            t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;

            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do {
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }

    //printf("[n=%d] num_steps=%d, near=%f, far=%f, dt=%f, max_steps=%f\n", n, num_steps, near, far, dt_min, (far - near) / dt_min);

    // second pass: really locate and write points & dirs
    uint32_t point_index = atomicAdd(counter, num_steps);
    uint32_t ray_index = atomicAdd(counter + 1, 1);

    //printf("[n=%d] num_steps=%d, point_index=%d, ray_index=%d\n", n, num_steps, point_index, ray_index);



    // write rays
    rays[ray_index * 3] = n;
    rays[ray_index * 3 + 1] = point_index;
    rays[ray_index * 3 + 2] = num_steps;

    if (num_steps == 0) return;
    if (point_index + num_steps > M) return;


    xyzs += point_index * 3;
    dirs += point_index * 3;
    deltas += point_index * 2;

    sun_dirs += point_index * 3;
    all_ts += point_index * 4;

    t = t0;
    uint32_t step = 0;

    float last_t = t;

    while (t < far && step < num_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;

        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        // query grid
        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;

            sun_dirs[0] = sdx;
            sun_dirs[1] = sdy;
            sun_dirs[2] = sdz;

            all_ts[0] = tsx;
            all_ts[1] = tsy;
            all_ts[2] = tsz;
            all_ts[3] = tsw;

            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            xyzs += 3;
            dirs += 3;
            deltas += 2;

            sun_dirs += 3;
            all_ts += 4;

            step++;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do {
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }
}


void march_rays_train(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor sun_d, const at::Tensor ts,
                    const at::Tensor grid, const float bound, const float dt_gamma, const uint32_t max_steps,
                    const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M, const at::Tensor nears, const at::Tensor fars,
                    at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor sun_dirs, at::Tensor all_ts, at::Tensor rays, at::Tensor counter, at::Tensor noises) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays_train", ([&] {
        kernel_march_rays_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(
            rays_o.data_ptr<scalar_t>(),
            rays_d.data_ptr<scalar_t>(),

            sun_d.data_ptr<scalar_t>(),
            ts.data_ptr<scalar_t>(),

            grid.data_ptr<uint8_t>(),
            bound, dt_gamma, max_steps, N, C, H, M,
            nears.data_ptr<scalar_t>(),
            fars.data_ptr<scalar_t>(),
            xyzs.data_ptr<scalar_t>(),
            dirs.data_ptr<scalar_t>(),
            deltas.data_ptr<scalar_t>(),

            sun_dirs.data_ptr<scalar_t>(),
            all_ts.data_ptr<scalar_t>(),

            rays.data_ptr<int>(),
            counter.data_ptr<int>(),
            noises.data_ptr<scalar_t>());
    }));
}

// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,
    const scalar_t * __restrict__ deltas,

    const scalar_t * __restrict__ sun_v,
    // const scalar_t * __restrict__ sky_color,
    // const scalar_t * __restrict__ irradiance,
    const scalar_t * __restrict__ beta,

    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * weights_sum,
    scalar_t * depth,
    scalar_t * image,

    scalar_t * shade, // shape (nb_rayons, nb_max_point) ex: 4096, 120
    scalar_t * uncert, // shape (nb_rayons, nb_max_point) ex: 4096, 120
    scalar_t * transparency, // shape (nb_rayons, nb_max_point) ex: 4096, 120
    scalar_t * opacity, // shape (nb_rayons, nb_max_point) ex: 4096, 120
    scalar_t * weights, // shape (nb_rayons, nb_max_point) ex: 4096, 120

    const uint32_t max_point
    // scalar_t *weights
    // scalar_t * w_beta
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;
    // printf("[n%d<=N%d]", n, N);

    // locate
    uint32_t index = rays[n * 3]; // index du rayon
    uint32_t offset = rays[n * 3 + 1]; // nombre de points samplés entre le rayons précedents et ce rayon
    uint32_t num_steps = rays[n * 3 + 2]; // nombre de points samplés sur ce rayon
    // if (num_steps >= max_point){printf("[num_steps %d<= max %d]", num_steps, max_point); }

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        // printf("num_steps == 0 || offset + num_steps > M ");

        weights_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;



        // // ugly solution but could be paralized
        uint32_t step = 0;
        while (step < max_point) {
            transparency[index * max_point + step] = 0;
            opacity[index * max_point + step] = 0;
            shade[index * max_point + step] = 0;
            uncert[index * max_point + step] = 0;
            weights[index * max_point + step] = 0;
            step++;
        }
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;

    sun_v += offset;
    // sky_color += offset * 3;
    beta += offset;

    // accumulate
    uint32_t step = 0;
    // scalar_t step_div = 0;
    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, t = 0, d = 0;

    scalar_t _term2 = 0.0f, _term3 = 0.0f, _beta = 0.0f;
    // scalar_t r_irra = 0, g_irra = 0, b_irra = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        // printf("\n[n=%d] sigmas=%d, deltas=%f\n", n, sigmas[0], deltas[0]);
        transparency[index * max_point + step] = T;
        opacity[index * max_point + step] = alpha;
        const scalar_t weight = alpha * T;

        shade[index * max_point + step] = sun_v[0];
        uncert[index * max_point + step] = beta[0];

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        t += deltas[1]; // real delta
        d += weight * t;

        ws += weight;

        T *= 1.0f - alpha;

        weights[index * max_point + step] = weight;
        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;

        sun_v++;
        beta++;

        step++;
    }


    // since empty init bring rand values >> set the rest to 0
    while (step < max_point) {
        transparency[index * max_point + step] = 0;
        opacity[index * max_point + step] = 0;
        shade[index * max_point + step] = 0;
        uncert[index * max_point + step] = 0;
        weights[index * max_point + step] = 0;
        step++;
    }

    // write
    weights_sum[index] = ws; // weights_sum
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;

    shade[index] = (_term2 + (1 -_term3)); //step_div;
    uncert[index] = _beta; //step_div;
}


void composite_rays_train_forward(const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas,
                                 const at::Tensor rays,const at::Tensor sun_v,const at::Tensor beta,
                                 const uint32_t M, const uint32_t N, const float T_thresh,
                                 at::Tensor weights_sum, at::Tensor depth, at::Tensor image,
                                 at::Tensor shade, at::Tensor uncert,
                                 at::Tensor transparency, at::Tensor opacity, at::Tensor weights, const uint32_t max_point) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(
            sigmas.data_ptr<scalar_t>(),
            rgbs.data_ptr<scalar_t>(),
            deltas.data_ptr<scalar_t>(),

            sun_v.data_ptr<scalar_t>(),
            // sky_color.data_ptr<scalar_t>(),
            // irradiance.data_ptr<scalar_t>(),
            beta.data_ptr<scalar_t>(),

            rays.data_ptr<int>(), M, N, T_thresh,
            weights_sum.data_ptr<scalar_t>(),
            depth.data_ptr<scalar_t>(),
            image.data_ptr<scalar_t>(),

            shade.data_ptr<scalar_t>(),
            uncert.data_ptr<scalar_t>(),

            transparency.data_ptr<scalar_t>(),
            opacity.data_ptr<scalar_t>(),
            weights.data_ptr<scalar_t>(),
            max_point
            // weights.data_ptr<scalar_t>()
            // w_beta.data_ptr<scalar_t>()
            );
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_backward(
    const scalar_t * __restrict__ grad_depth,
    const scalar_t * __restrict__ grad_opacity,
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,

    const scalar_t * __restrict__ grad_uncert,
    const scalar_t * __restrict__ grad_shade,
    const scalar_t * __restrict__ betas,
    const scalar_t * __restrict__ sun_v,

    const scalar_t * __restrict__ deltas,

    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ depth,
    const scalar_t * __restrict__ opacity,
    const scalar_t * __restrict__ image,
    const scalar_t * __restrict__ uncert,
    const scalar_t * __restrict__ shade,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs,

    scalar_t * grad_sun_v,
    scalar_t * grad_betas
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    // printf("[index=%d], [offset=%d], [num_steps=%d]\n", index, offset, num_steps);

    // TEMPLATE for vals net outputs
    // grad_x_from_loss += index * dim; // val grad from loss
    // final_x_val_predicted += index * dim; // val predict pixel color
    // x_val_predicted_along_the_ray += offset * dim; // color val predicted for each sample
    // grad_foreach_x_val_predicted_along_the_ray += offset * dim; // grad propagated to the network for autograd

    // for rgbs net outputs
    grad_image += index * 3; // val grad from loss
    image += index * 3; // val predict pixel color
    rgbs += offset * 3; // color val predicted for each sample
    grad_rgbs += offset * 3; // grad propagated to the network for autograd

    // for sigmas net outputs
    grad_weights_sum += index; // val grad from loss for sigma
    weights_sum += index; // val predict for final pixel color
    sigmas += offset; // density val predicted for each sample
    deltas += offset * 2; // val position for each sample along ray
    grad_sigmas += offset; // grad propagated to the network for autograd


    // for beta net outputs
    grad_uncert += index; // beta val grad from loss
    uncert += index; // val predict pixel color
    betas += offset; // color val predicted for each sample
    grad_betas += offset; // grad propagated to the network for autograd


    // for sun_v net outputs
    grad_shade += index; // beta val grad from loss
    shade += index; // shade val predict
    sun_v += offset; // direct light val predicted for each sample
    grad_sun_v += offset; // grad propagated to the network for autograd


    grad_opacity += index; // val grad from loss for sigma
    opacity += index; // final opacity val


    grad_depth += index;
    depth += index;


    // accumulate
    uint32_t step = 0;

    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0], opacity_final = opacity[0], beta_final = uncert[0], shade_final = shade[0], d_final = depth[0];
    scalar_t r = 0, g = 0, b = 0, ws = 0, t = 0, d =0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        ws += weight;

        t += deltas[1]; // real delta
        d += weight * t;

        T *= 1.0f - alpha;
  
        grad_betas[0] =  deltas[0] * grad_uncert[0] * betas[0];
        grad_sun_v[0] = deltas[0] * grad_shade[0] * sun_v[0];

        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;

        grad_sigmas[0] = deltas[0] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) +
            grad_image[1] * (T * rgbs[1] - (g_final - g)) +
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +  // gradients from rgb
            grad_opacity[0]*(1-opacity_final) + // gradient from opacity
            grad_weights_sum[0] * (1 - ws_final) // gradient from ws
        );




        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        betas++; // avant pas fais (forgot) mais ça marhe aussi et surtout plus rapide !
        sun_v++; // du coup à voir si ici pareil


        grad_sigmas++;
        grad_rgbs += 3;

        grad_betas++;
        grad_sun_v++;


        step++;
    }
}


void composite_rays_train_backward(const at::Tensor grad_depth, const at::Tensor grad_opacity, const at::Tensor grad_weights_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs,
                                    const at::Tensor grad_uncert, const at::Tensor grad_shade, const at::Tensor beta, const at::Tensor sun_v,
                                    const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor depth, const at::Tensor opacity,
                                    const at::Tensor image, const at::Tensor uncert, const at::Tensor shade, const uint32_t M, const uint32_t N, const float T_thresh,
                                     at::Tensor grad_sigmas, at::Tensor grad_rgbs, at::Tensor grad_sun_v, at::Tensor grad_betas) {

    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(
            grad_depth.data_ptr<scalar_t>(),
            grad_opacity.data_ptr<scalar_t>(),
            grad_weights_sum.data_ptr<scalar_t>(),
            grad_image.data_ptr<scalar_t>(),
            sigmas.data_ptr<scalar_t>(),
            rgbs.data_ptr<scalar_t>(),

            grad_uncert.data_ptr<scalar_t>(),
            grad_shade.data_ptr<scalar_t>(),
            beta.data_ptr<scalar_t>(),
            sun_v.data_ptr<scalar_t>(),

            deltas.data_ptr<scalar_t>(),
            rays.data_ptr<int>(),
            weights_sum.data_ptr<scalar_t>(),
            opacity.data_ptr<scalar_t>(),
            depth.data_ptr<scalar_t>(),
            image.data_ptr<scalar_t>(),
            uncert.data_ptr<scalar_t>(),
            shade.data_ptr<scalar_t>(), M, N, T_thresh,
            grad_sigmas.data_ptr<scalar_t>(),
            grad_rgbs.data_ptr<scalar_t>(),

            grad_sun_v.data_ptr<scalar_t>(),
            grad_betas.data_ptr<scalar_t>()
            );
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////

template <typename scalar_t>
__global__ void kernel_march_rays(
    const uint32_t n_alive,
    const uint32_t n_step,
    const int* __restrict__ rays_alive,
    const scalar_t* __restrict__ rays_t,
    const scalar_t* __restrict__ rays_o,
    const scalar_t* __restrict__ rays_d,

    const scalar_t* __restrict__ sun_d,
    const scalar_t * __restrict__ ts,


    const float bound,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t C, const uint32_t H,
    const uint8_t * __restrict__ grid,
    const scalar_t* __restrict__ nears,
    const scalar_t* __restrict__ fars,
    scalar_t* xyzs, scalar_t* dirs, scalar_t* deltas,

    scalar_t* sun_dirs,
    scalar_t * all_ts,

    const scalar_t* __restrict__ noises
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    const float noise = noises[n];

    // locate
    rays_o += index * 3;
    rays_d += index * 3;

    sun_d += index * 3;
    ts += n * 4;

    xyzs += n * n_step * 3;
    dirs += n * n_step * 3;
    deltas += n * n_step * 2;

    sun_dirs += n * n_step * 3;
    all_ts += n * n_step * 4;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];

    const float sdx = sun_d[0], sdy = sun_d[1], sdz = sun_d[2];
    const float tsx = ts[0], tsy = ts[1], tsz = ts[2], tsw = ts[3];

    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;

    float t = rays_t[index]; // current ray's t
    // const float near = nears[index], far = fars[index];
    const float far = fars[index];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;

    // march for n_step steps, record points
    uint32_t step = 0;

    // introduce some randomness
    t += clamp(t * dt_gamma, dt_min, dt_max) * noise;

    float last_t = t;

    while (t < far && step < n_step) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1, level), bound);
        const float mip_rbound = 1 / mip_bound;

        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;

            sun_dirs[0] = sdx;
            sun_dirs[1] = sdy;
            sun_dirs[2] = sdz;

            all_ts[0] = tsx;
            all_ts[1] = tsy;
            all_ts[2] = tsz;
            all_ts[3] = tsw;

            // recup ts ici ts[0] = t

            // calc dt
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            // step
            xyzs += 3;
            dirs += 3;
            deltas += 2;

            sun_dirs += 3;
            all_ts += 4;

            step++;

        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do {
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }
}


void march_rays(const uint32_t n_alive, const uint32_t n_step, const at::Tensor rays_alive, const at::Tensor rays_t,
                const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor sun_d, const at::Tensor ts, const float bound,
                const float dt_gamma, const uint32_t max_steps, const uint32_t C, const uint32_t H,
                const at::Tensor grid, const at::Tensor near, const at::Tensor far,
                at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor sun_dirs, at::Tensor all_ts, at::Tensor noises) {
    static constexpr uint32_t N_THREAD = 256;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays", ([&] {
        kernel_march_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(
            n_alive, n_step,
            rays_alive.data_ptr<int>(),
            rays_t.data_ptr<scalar_t>(),
            rays_o.data_ptr<scalar_t>(),
            rays_d.data_ptr<scalar_t>(),

            sun_d.data_ptr<scalar_t>(),
            ts.data_ptr<scalar_t>(),

            bound, dt_gamma, max_steps, C, H,
            grid.data_ptr<uint8_t>(),
            near.data_ptr<scalar_t>(),
            far.data_ptr<scalar_t>(),
            xyzs.data_ptr<scalar_t>(),
            dirs.data_ptr<scalar_t>(),
            deltas.data_ptr<scalar_t>(),

            sun_dirs.data_ptr<scalar_t>(),
            all_ts.data_ptr<scalar_t>(),

            noises.data_ptr<scalar_t>());
    }));
}


template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive,
    const uint32_t n_step,
    const float T_thresh,
    int* rays_alive,
    scalar_t* rays_t,
    const scalar_t* __restrict__ sigmas,
    const scalar_t* __restrict__ rgbs,
    const scalar_t* __restrict__ beta,
    const scalar_t* __restrict__ deltas,

    // const scalar_t * __restrict__ sun_sc,
    // const scalar_t * __restrict__ sky_color,
    // const scalar_t * __restrict__ irradiance,

    scalar_t* weights_sum, scalar_t* depth, scalar_t* image, scalar_t * uncert_scene

    // scalar_t* uncert, scalar_t* shade
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id

    // locate
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;

    beta += n * n_step;
    // sky_color += n * n_step * 3;
    // irradiance += n * n_step * 3;


    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;

    uncert_scene += index;
    // shade += index;

    scalar_t t = rays_t[0]; // current ray's t

    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t us = uncert_scene[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];

    // scalar_t _uncert = uncert[0];
    // scalar_t _shade = shade[0];

    // scalar_t _term2 = 0.0f, _term3 = 0.0f, _beta = 0.0f, _sun = 0.0f;
    // accumulate
    uint32_t step = 0;
    while (step < n_step) {

        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        // temp = T*T -2*T*sun_sc[0] + sun_sc[0]*sun_sc[0];
        // _term2 += temp; // = Somme (T_i - s_i)**2
        // _term3 += weight * sun_sc[0];

        t += deltas[1]; // real delta
        d += weight * t;
        us += weight * beta[0];

        // printf("[n=%d] rgb=(%f, %f, %f), irradiance=(%f, %f, %f), d=%f\n", n, rgbs[0], rgbs[1], rgbs[2], irradiance[0], irradiance[1], irradiance[2], d);
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;

        beta++;
        // irradiance += 3;

        step++;
    }

    // if (n==1024){
    //     printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);
    // }


    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;

    uncert_scene[0] = us;
    // term3[0] = _term3;
}


void composite_rays(const uint32_t n_alive, const uint32_t n_step, const float T_thresh,
                    at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor beta,
                    at::Tensor deltas,
                    at::Tensor weights, at::Tensor depth, at::Tensor image, at::Tensor uncert_scene) {

    static constexpr uint32_t N_THREAD = 256;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(
            n_alive, n_step, T_thresh,
            rays_alive.data_ptr<int>(),
            rays_t.data_ptr<scalar_t>(),
            sigmas.data_ptr<scalar_t>(),
            rgbs.data_ptr<scalar_t>(),
            beta.data_ptr<scalar_t>(),
            deltas.data_ptr<scalar_t>(),

            weights.data_ptr<scalar_t>(),
            depth.data_ptr<scalar_t>(),
            image.data_ptr<scalar_t>(),
            uncert_scene.data_ptr<scalar_t>());
    }));
}
